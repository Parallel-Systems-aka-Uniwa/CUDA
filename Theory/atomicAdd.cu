/* Εύρεση του αθροίσματος ενός μεγάλου πλήθους αριθμών με χρήση δεντρικού
αλγόριθμου και χρήση της ατομικής εντολής atomicAdd() */
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define nThreads 1024
#define nBlocks (int)ceil((float)N/nThreads)

__global__ void add1( float *R, float *C) 
{
    __shared__ float smarray[nThreads];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int smindex = threadIdx.x;
    // Μεταφορά στην κοινή μνήμη των αριθμών που αντιστοιχούν στο μπλοκ
    smarray[smindex] = R[tid];
    // Αναμονή των νημάτων του μπλοκ μέχρι να ολοκληρωθεί η μεταφορά
    __syncthreads();
    // Εφαρμογή δεντρικού αλγόριθμου για τον υπολογισμό του αθροίσματος
    int i = blockDim.x/2;
    while (i != 0) 
    {
        if (smindex < i)
            smarray[smindex] += smarray[smindex + i];
        __syncthreads();
        i /= 2;
    }
    
    // Ενημέρωση του καθολικού αθροίσματος με χρήση ατομικής εντολής
    if (smindex == 0)
        atomicAdd(C, smarray[0]);
    }

int main(void) 
{
    float *R, *C;
    float *Rd, *Cd;
    
    // Εκχώρηση μνήμης στις δομές και μεταβλητές του host
    R = (float*)malloc( N*sizeof(float) );
    C = (float*)malloc( sizeof(float) );
    
    hipEvent_t start,stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Εκχώρηση μνήμης στις δομές και μεταβλητές της συσκευής
    hipMalloc( (void**)&Rd, N*sizeof(float) );
    hipMalloc( (void**)&Cd, sizeof(float) );

    // Αρχικοποίηση του πίνακα εισόδου στον host
    for (int i=0; i<N; i++) 
        R[i] = 1.0;

    // Μεταφορά των δεδομένων του πίνακα εισόδου από τον host στη συσκευή
    hipMemcpy(Rd, R, N*sizeof(float), hipMemcpyHostToDevice );
    hipEventRecord(start,0);
    
    add1<<<nBlocks,nThreads>>>(Rd, Cd);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    // Μεταφορά του αποτελέσματος από τη συσκευή στον host
    hipMemcpy(C, Cd, sizeof(float), hipMemcpyDeviceToHost);
    printf("Final sum is equal to %f \n", C[0]);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf ("Time for the kernel: %f ms\n", elapsedTime);

    // Αποδέσμευση μνήμης στη συσκευή
    hipFree(Rd );
    hipFree(Cd);

    // Αποδέσμευση μνήμης στον host
    free(R);
    free(C);
}