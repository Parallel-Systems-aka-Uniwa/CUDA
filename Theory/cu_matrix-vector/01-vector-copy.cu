#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[])
{
	int		N, i;
	int		*input_h, *output_h;	/* Pointers for vectors on the host. */
	int		*vector_d;		/* Pointer for vector on the device. */
	hipError_t	err;

	if (argc != 2) {
		printf("Usage:\n");
		printf("01-vector-copy <Size of vectors>\n");
		exit(1);
	}

	N = atoi(argv[1]);

	if (N < 1) {
		printf("Size of vectors should be at least 1.\n");
		exit(1);
	}

	if (N > 1024) {
		printf("Maximum size of vectors is 1024.\n");
		printf("Please be patient, we will get to larger problems soon!\n");
		exit(1);
	}

	/*
	 * Allocate memory for input and output vector on host.
	 */
	input_h = (int *)malloc(N * sizeof(int));
	if (input_h == NULL) {
		printf("Could not allocate memory for input vector on host.\n");
		exit(1);
	}

	output_h = (int *)malloc(N * sizeof(int));
	if (output_h == NULL) {
		printf("Could not allocate memory for output vector on host.\n");
		exit(1);
	}

	/*
	 * Initialize input and output vector on host.
	 * Notice that all elements of the output vector are initialized to zero.
	 */
	for (i = 0; i < N; i++) {
		input_h[i] = 1;
		output_h[i] = 0;
	}

	/*
	 * Here you will have to:
	 *  - Allocate memory for a vector on the device.
	 *  - Copy the input vector from the host to the device.
	 *  - Copy the vector from the device to the output vector on the host.
	 */
	err = hipMalloc(&vector_d, N * sizeof(int));
	if (err != hipSuccess) {
		printf("Could not allocate memory for vector on the device.\n");
                exit(1);
        }

	err = hipMemcpy(vector_d, input_h, N * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
		printf("Could not copy input vector to device.\n");
		exit(1);
        }

	err = hipMemcpy(output_h, vector_d, N * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
		printf("Could not copy vector from device to output vector on host.\n");
		exit(1);
        }

	/*
	 * If you completed the previous tasks correctly,
	 * all elements of the output vector should now contain one.
	 * We print a few of them to make certain that everything went fine.
	 */

	printf("output_h[%4d] = %d\n", 0, output_h[0]);
	printf("output_h[%4d] = %d\n", (N - 1) / 2, output_h[(N - 1) / 2]);
	printf("output_h[%4d] = %d\n", N - 1, output_h[N - 1]);
}


