/* Εύρεση του αθροίσματος ενός μεγάλου πλήθους αριθμών με χρήση δεντρικού
αλγόριθμου και επιπλέον πυρήνα/kernel για συγχρονισμό */
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define nThreads 1024
#define nBlocks (int)ceil((float)N/nThreads)

// Η συνάρτηση πυρήνα για την πρώτη εργασία μείωσης (1ος kernel)
__global__ void add1( float *R, float *tempC) 
{
    __shared__ float smarray[nThreads];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int smindex = threadIdx.x;
    // Μεταφορά στην κοινή μνήμη των αριθμών που αντιστοιχούν στο μπλοκ
    smarray[smindex] = R[tid];
    // Αναμονή των νημάτων του μπλοκ μέχρι να ολοκληρωθεί η μεταφορά
    __syncthreads();
    // Εφαρμογή δεντρικού αλγόριθμου για τον υπολογισμό του αθροίσματος
    int i = blockDim.x/2;
    
    while (i != 0) 
    {
        if (smindex < i)
        smarray[smindex] += smarray[smindex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (smindex == 0)
        tempC[blockIdx.x] = smarray[0];
}

// Η συνάρτηση πυρήνα για τη δεύτερη εργασία μείωσης (2ος kernel)
__global__ void add2(float *tempC, float *C) 
{
    __shared__ float smarray[nThreads];
    int tid = threadIdx.x;
    int smindex = threadIdx.x;
    
    // Μεταφορά στην κοινή μνήμη του μπλοκ των αριθμών που θα προστεθούν
    smarray[smindex] = tempC[tid];
    // Αναμονή των νημάτων του μπλοκ μέχρι να ολοκληρωθεί η μεταφορά
    __syncthreads();
    // Εφαρμογή δεντρικού αλγόριθμου για τον υπολογισμό του αθροίσματος
    int i = blockDim.x/2;
    while (i != 0) 
    {
        if (smindex < i)
        smarray[smindex] += smarray[smindex + i];
        __syncthreads();
        i /= 2;
    }

    if (smindex == 0)
        C[0] = smarray[0];
}

int main(void) 
{
    float *R, *C;
    float *Rd, *tempCd, *Cd;
    
    // Εκχώρηση μνήμης στις δομές και μεταβλητές του host
    R = (float*)malloc( N*sizeof(float) );
    C = (float*)malloc( sizeof(float) );

    hipEvent_t start,stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Εκχώρηση μνήμης στις δομές και μεταβλητές της συσκευής
    hipMalloc( (void**)&Rd, N*sizeof(float) );
    hipMalloc( (void**)&tempCd, nBlocks*sizeof(float) );
    hipMalloc( (void**)&Cd, sizeof(float) );
    
    // Αρχικοποίηση του πίνακα εισόδου στον host
    for (int i=0; i<N; i++) 
        R[i] = 1.0;
    
    // Μεταφορά των δεδομένων του πίνακα εισόδου από τον host στη συσκευή
    hipMemcpy(Rd, R, N*sizeof(float), hipMemcpyHostToDevice );
    hipEventRecord(start,0);
    
    // Κλήση-εκτέλεση του 1ου kernel
    add1<<<nBlocks,nThreads>>>(Rd, tempCd);
    // Κλήση-εκτέλεση του 2ου kernel
    add2<<<1,nThreads>>>(tempCd, Cd);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    // Μεταφορά του αποτελέσματος από τη συσκευή στον host
    hipMemcpy(C, Cd, sizeof(float), hipMemcpyDeviceToHost);
    printf("Final sum is equal to %f \n", C[0]);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf ("Time for the kernel: %f ms\n", elapsedTime);
    
    // Αποδέσμευση μνήμης στη συσκευή
    hipFree(Rd );
    hipFree(tempCd);
    hipFree(Cd);
    
    // Αποδέσμευση μνήμης στον host
    free(R);
    free(C);
}