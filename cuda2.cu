/*
 *  === Αρχείο: cuda2.cu ===
 *
 *  Ονοματεπώνυμο: Αθανασίου Βασίλειος Ευάγγελος
 *  Αριθμός Μητρώου: 19390005
 *  Πρόγραμμα Σπουδών: ΠΑΔΑ
 *  
 *  Μεταγλώττιση: nvcc -o cuda2 cuda2.cu
 *  Εκτέλεση: ./cuda2 A.txt A_cov.txt
 * 
 */
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 10
#define BL 5
#define T 2

int main(int argc, char *argv[])
{
    int **A, **A_cov;
    int *d_A;
    FILE *fp_A, *fp_A_cov;
    int i, j;
    int matrix_size, grid_size, block_size;

    matrix_size = N;
    grid_size = BL;
    block_size = T;
    
    if (block_size < 1 || block_size > 1024)
    {
        printf("Threads per block must be between 1 to 1024.\n");
        exit(1);
    }

    if (grid_size < 1 || grid_size > 65535)
    {
        printf("Blocks must be between 1 to 65535.\n");
        exit(1);
    }

    if (argc != 3) 
    {
        printf("Usage: %s A.txt A_cov.txt\n", argv[0]);
        exit(1);
    }

    fpA = fopen(argv[1], "w");
    if (fpA == NULL) 
    {
        printf("Cannot open file %s\n", argv[1]);
        exit(1);
    }

    fpA_cov = fopen(argv[2], "w");
    if (fpA_cov == NULL) 
    {
        printf("Cannot open file %s\n", argv[2]);
        exit(1);
    }

    printf("Matrix size  : %d x %d\n", matrix_size, matrix_size );
    printf("Grid size    : %d\n", grid_size);
    printf("Block size   : %d\n", block_size);
    printf("-----------------------------------\n");

    A = (int **) malloc(matrix_size * sizeof(int *));
    A_cov = (int **) malloc(matrix_size * sizeof(int *));
    for (i = 0; i < matrix_size; i++) 
    {
        A[i] = (int *) malloc(matrix_size * sizeof(int));
        if (A[i] == NULL) 
        {
            printf("Memory allocation failed for A[%d]\n", i);
            exit(1);
        }
        A_cov[i] = (int *) malloc(matrix_size * sizeof(int));
        if (A_cov[i] == NULL) 
        {
            printf("Memory allocation failed for A_cov[%d]\n", i);
            exit(1);
        }
    }

    srand(time(NULL));

    for (i = 0; i < size; i++)
    {
        for (j = 0; j < size; j++)
        {
            A[i][j] = rand() % 199 - 99;                           // Τιμές στο διάστημα [-99, 99]
            A[i][j] = A[i][j] >= 0 ? A[i][j] + 10 : A[i][j] - 10;  // Τυχαία επιλογή προσήμου
        }
    }


    return 0;
}