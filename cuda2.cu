#include "hip/hip_runtime.h"
/*
 *  === Αρχείο: cuda2.cu ===
 *
 *  Ονοματεπώνυμο: Αθανασίου Βασίλειος Ευάγγελος
 *  Αριθμός Μητρώου: 19390005
 *  Πρόγραμμα Σπουδών: ΠΑΔΑ
 *  
 *  Μεταγλώττιση: nvcc -o cuda2 cuda2.cu
 *  Εκτέλεση: ./cuda2 A.txt A_cov.txt
 * 
 */
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 8
#define nThreads 4
#define nBlocks (int)ceil((float)N/nThreads)

int main(int argc, char *argv[])
{
    int *h_A;
    int *d_A;

    int n, threadsPerBlock, blocksPerGrid;
    int intbytes;
    int max_threads, max_block_dimX, max_block_dimY, max_block_dimZ, max_grid_dimX, max_grid_dimY, max_grid_dimZ;
    int i, j;
    FILE *fpA, *fpAcov;
    float elapsedTime;

    hipEvent_t start, stop;
    hipError_t err;
    hipDeviceProp_t prop;

    if (argc != 3)
    {
        printf("Usage: %s A.txt A_cov.txt\n", argv[0]);
        exit(1);
    }

    n = N;
    threadsPerBlock = nThreads;
    blocksPerGrid = nBlocks;

    hipGetDeviceProperties(&prop, 0); // 0 is the device ID

    max_threads = prop.maxThreadsPerBlock;
    max_block_dimX = prop.maxThreadsDim[0];
    max_block_dimY = prop.maxThreadsDim[1];
    max_block_dimZ = prop.maxThreadsDim[2];
    max_grid_dimX = prop.maxGridSize[0];
    max_grid_dimY = prop.maxGridSize[1];
    max_grid_dimZ = prop.maxGridSize[2];

    printf("--------------- Device Properties ---------------\n");
    printf("Device name           : %s\n", prop.name);
    printf("Max threads per block : %d\n", max_threads);
    printf("Max block dimensions  : %d x %d x %d\n", max_block_dimX, max_block_dimY, max_block_dimZ);
    printf("Max grid dimensions   : %d x %d x %d\n", max_grid_dimX, max_grid_dimY, max_grid_dimZ);
    printf("-------------------------------------------------\n");

    if (n < 1)
    { printf("Error --> Matrix size must be at least 1\n"); exit(1); }
    if (threadsPerBlock < 1)
    { printf("Error --> Threads per block (block size) must be at least 1\n"); exit(1); }
    if (blocksPerGrid < 1)
    { printf("Error --> Blocks per grid (grid size) must be at least 1\n"); exit(1); }
    if (threadsPerBlock > max_threads)
    { printf("Error --> Threads per block (block size) exceed maximum allowed for %s\n", prop.name); exit(1); }
    if (blocksPerGrid > max_grid_dimX)
    { printf("Error --> Blocks per grid (grid size) exceed maximum allowed for %s\n", prop.name); exit(1); }

    fpA = fopen(argv[1], "w");
    if (fpA == NULL) { printf("Cannot open file %s\n", argv[1]); exit(1); }
    fpAcov = fopen(argv[2], "w");
    if (fpAcov == NULL) { printf("Cannot open file %s\n", argv[2]); exit(1); }

    err = hipEventCreate(&start);
    if (err != hipSuccess) { printf("CUDA Error --> hipEventCreate(&start) failed.\n"); exit(1); }
    err = hipEventCreate(&stop);
    if (err != hipSuccess) { printf("CUDA Error --> hipEventCreate(&stop) failed.\n"); exit(1); }
  
    printf("--------------- Input Parameters ---------------\n");
    printf("Matrix size        : %d x %d\n", n, n);
    printf("Blocks per Grid    : %d\n", blocksPerGrid);
    printf("Threads per Block  : %d\n", threadsPerBlock);
    printf("------------------------------------------------\n");

    intBytes = n * n * sizeof(int);

    h_A = (int *) malloc(intBytes);
    if (h_A == NULL) { printf("Error --> Memory allocation failed for A.\n"); exit(1); }

    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++)
        {
            h_A[i * n + j] = rand() % 199 - 99;                           // Τιμές στο διάστημα [-99, 99]
            h_A[i * n + j] = h_A[i * n + j] >= 0 ? h_A[i * n + j] + 10 : h_A[i * n + j] - 10;  // Τυχαία επιλογή προσήμου
        }
    
    err = hipMalloc((void **) &d_A, intBytes);
    if (err != hipSuccess) { printf("CUDA Error --> hipMalloc((void **) &d_A, bytes) failed."); exit(1); }

    err = hipMemcpy(d_A, h_A, intBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("CUDA Error --> hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice) failed."); exit(1); }

    dim3 dimBlock(nThreads, nThreads);
    dim3 dimGrid(nBlocks, nBlocks);

    hipEventRecord(start, 0);

    // Κλήση του kernel

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time for the kernel: %f ms\n", elapsedTime);

    free(h_A);
    hipFree(d_A);

    fclose(fpA);
    fclose(fpAcov);

    return 0;
}